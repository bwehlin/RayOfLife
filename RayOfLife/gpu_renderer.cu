#include "hip/hip_runtime.h"
﻿#include "gpu_renderer.cuh"
#include "support.cuh"
#include <iostream>
#include <hip/hip_runtime.h>
#include <>
#include <device_atomic_functions.h>

#include "vector_math.cuh"
#include "amantides_woo.cuh"
#include "intersect.cuh"
#include "game.cuh"

namespace
{
  __device__ bool isAlive(bool* game, int x, int y, int z, int cellsPerDim)
  {
    return game[z * cellsPerDim * cellsPerDim + y * cellsPerDim + x];
  }

  __device__ void castRay(rol::RayIntersection& intersection, 
    rol::AmantidesWooState& awstate, 
    fptype3 rayOrigin, fptype3 rayDirection, 
    bool* game, int cellsPerDim, rol::SceneData* scene, fptype3 cameraOrigin)
  {
    while (true)
    {
      if (awstate.pos.x < 0 || awstate.pos.x >= cellsPerDim
        || awstate.pos.y < 0 || awstate.pos.y >= cellsPerDim
        || awstate.pos.z < 0 || awstate.pos.z >= cellsPerDim)
      {
        // We have fallen out of the cell grid
        intersection.hit = false;
        return;
      }

      if (isAlive(game, awstate.pos.x, awstate.pos.y, awstate.pos.z, cellsPerDim))
      {
        intersection = rol::traceRay(rayOrigin, rayDirection, awstate.pos, *scene, cameraOrigin);
        if (intersection.hit)
        {
          return;
        }
      }

      nextAwStep(awstate);
    }

    intersection.hit = false;
  }
  
  __device__ fptype3 subpixelColor(fptype x, fptype y, fptype3 cameraOrigin, 
    int cellsPerDim, int depth, bool* game, rol::SceneData* scene)
  {
    auto rayOrigin = cameraOrigin;
    auto cameraTarget = makeFp3(cameraOrigin.x + 1.f, x, y);

    auto rayDirection = normalize(cameraTarget - rayOrigin);

    auto color = makeFp3(0.f, 0.f, 0.f);
    fptype reflection = 1.f;

    rol::AmantidesWooState awstate;
    rol::initAmantidesWoo(awstate, rayOrigin, rayDirection, cellsPerDim);
    if (awstate.pos.x != 0)
    {
      // Ray from origin does not hit cell grid
      color = makeFp3(1.f, 1.f, 0.f);
      return color;
    }

    rol::RayIntersection intersection;
    while (depth--)
    {
      castRay(intersection, awstate, rayOrigin, rayDirection, game, cellsPerDim, scene, cameraOrigin);
      if (!intersection.hit)
      {
        break;
      }

      rayOrigin = intersection.point + intersection.normal * static_cast<fptype>(0.0001f);
      rayDirection = normalize(rayDirection - 2 * dot(rayDirection, intersection.normal) * intersection.normal);

      rol::initAmantidesWooInside(awstate, rayOrigin, rayDirection, cellsPerDim);
      rol::nextAwStep(awstate);

      color += reflection * intersection.color;
      reflection *= scene->sphereReflection;
    }

    return color;
  }

  __global__ void renderSubpixels(fptype3* image, int w, int h,
    fptype2 screenMin, fptype2 screenMax, int subpixels, int maxDepth,
    bool* game, int cellsPerDim,
    fptype3 cameraOrigin, rol::SceneData* scene)
  {
    auto ix = blockIdx.x * blockDim.x + threadIdx.x;
    auto iy = blockIdx.y * blockDim.y + threadIdx.y;
    
    auto x = screenMin.x + (screenMax.x - screenMin.x) * ix / fptype(w * subpixels);
    auto y = screenMin.y + (screenMax.y - screenMin.y) * iy / fptype(h * subpixels);
    
    int imx = ix / subpixels;
    int imy = iy / subpixels;

    auto imoffset = imy * w + imx;

    auto color = subpixelColor(x, y, cameraOrigin, cellsPerDim, maxDepth, game, scene);
    atomicAdd(&image[imoffset].x, color.x);
    atomicAdd(&image[imoffset].y, color.y);
    atomicAdd(&image[imoffset].z, color.z);
  }

  __global__ void normalizePixels(fptype3* image, int w, int subpixels)
  {
    auto ix = blockIdx.x * blockDim.x + threadIdx.x;
    auto iy = blockIdx.y * blockDim.y + threadIdx.y;

    auto offset = iy * w + ix;
    auto factor = fptype{ 1.f } / fptype{ subpixels };
    factor *= factor;

    image[offset].x *= factor;
    image[offset].y *= factor;
    image[offset].z *= factor;
  }

}

rol::GpuRenderer::GpuRenderer(size_t w, size_t h)
  : Renderer(w, h)
  , m_d_game(nullptr)
  , m_imageData(nullptr)
  , m_d_subpixelBuffer(nullptr)
{
  CHK_ERR(hipMallocManaged(&m_imageData, sizeof(fptype3) * w * h))
  CHK_ERR(hipMallocManaged(&m_scene, sizeof(rol::SceneData)))

  *m_scene = SceneData();
}

rol::GpuRenderer::~GpuRenderer()
{
  auto freeMem = [](void* ptr)
  {
    if (!ptr)
    {
      return;
    }
    auto err = hipFree(ptr);
    if (err != hipSuccess)
    {
      std::cerr << "Warning! Could not free memory while destroying GPU renderer: " << hipGetErrorString(err) << '\n';
    }
  };
  
  freeMem(m_imageData);
  freeMem(m_d_game);
  freeMem(m_scene);
  freeMem(m_d_subpixelBuffer);
}

void rol::GpuRenderer::produceFrame(const Game& game, const Camera& camera,
  const fptype2& screenMin, const fptype2& screenMax)
{
  transferGameToGpu(game);

  for (int i = 0; i < width() * height(); ++i)
  {
    m_imageData[i] = makeFp3(0, 0, 0);
  }

  int blockDim = 16;
  if (width() % blockDim != 0 || height() % blockDim != 0
    || width() * subpixelCount() % blockDim != 0 || height() * subpixelCount() % blockDim != 0)
  {
    // Don't want to deal with block misalignment in the kernel if we don't absolutely have to.
    throw std::runtime_error("Screen dimensions and subpixel expansions must be multiples of " + std::to_string(blockDim));
  }

  auto subpixelBlocks = dim3(width() * subpixelCount() / blockDim, height() * subpixelCount() / blockDim);
  auto subpixelThreadsPerBlock = dim3(blockDim, blockDim);

  std::cout << "spb " << subpixelBlocks.x << " " << subpixelBlocks.y << std::endl;

  renderSubpixels<<<subpixelBlocks, subpixelThreadsPerBlock >>>(
    m_imageData, width(), height(),
    screenMin, screenMax, subpixelCount(), maxDepth(),
    m_d_game, game.cellsPerDim(),
    camera.origin, m_scene);

  auto err = hipGetLastError();
  if (err != hipSuccess)
  {
    throw CudaError(err, __FILE__, __LINE__);
  }

  auto pixelBlocks = dim3(width() / blockDim, height() / blockDim);
  auto pixelThreadsPerBlock = dim3(blockDim, blockDim);

  normalizePixels<<<pixelBlocks, pixelThreadsPerBlock>>>(m_imageData, width(), subpixelCount());
  err = hipGetLastError();
  if (err != hipSuccess)
  {
    throw CudaError(err, __FILE__, __LINE__);
  }

  CHK_ERR(hipDeviceSynchronize())

}

void
rol::GpuRenderer::transferGameToGpu(const Game& game)
{
  if (!m_d_game)
  {
    CHK_ERR(hipMalloc(&m_d_game, sizeof(bool) * game.cellsPerDim() * game.cellsPerDim() * game.cellsPerDim()));
  }

  auto nCellsPerDim = game.cellsPerDim();

  if (!m_h_game)
  {
    m_h_game = std::unique_ptr<bool[]>(new bool[nCellsPerDim * nCellsPerDim * nCellsPerDim]);
  }
  for (auto z = 0; z < nCellsPerDim; ++z)
  {
    for (auto y = 0; y < nCellsPerDim; ++y)
    {
      for (auto x = 0; x < nCellsPerDim; ++x)
      {
        m_h_game[z * nCellsPerDim * nCellsPerDim + y * nCellsPerDim + x] = game.isAlive(x, y, z);
      }
    }
  }

  CHK_ERR(hipMemcpy(m_d_game, m_h_game.get(), sizeof(bool) * nCellsPerDim * nCellsPerDim * nCellsPerDim, hipMemcpyHostToDevice))
}

const fptype3* rol::GpuRenderer::imageData() const
{
  return m_imageData;
}
