#include "hip/hip_runtime.h"
﻿#include "intersect.cuh"
#include "support.cuh"
#include "vector_math.cuh"

#include <cuda/std/utility>
#include <cuda/std/limits>

__host__ __device__
fptype intersectSphere(fptype3 rayOrigin, fptype3 rayDirection, const rol::SphereData& sphere)
{
  auto sphereCenter = sphere.position;
  auto sphereRadius = sphere.radius;

  auto a = dot(rayDirection, rayDirection);
  auto os = rayOrigin - sphereCenter;
  auto b = 2.f * dot(rayDirection, os);
  auto c = dot(os, os) - sphereRadius * sphereRadius;
  auto disc = b * b - 4 * a * c;
  if (disc > 0.f)
  {
    auto distSqrt = sqrt(disc);
    auto q = b < 0. ? (-b - distSqrt) / 2.f : (-b + distSqrt) / 2.f;
    auto t0 = q / a;
    auto t1 = c / q;
    if (t0 > t1)
    {
      cuda::std::swap(t0, t1);
    }
    if (t1 >= 0.)
    {
      return (t0 < 0.) ? t1 : t0;
    }
  }
  return cuda::std::numeric_limits<fptype>::infinity();
}

__host__ __device__
fptype intersectPlane(fptype3 rayOrigin, fptype3 rayDirection, const rol::PlaneData& plane)
{
  auto denom = dot(rayDirection, plane.normal);
  if (abs(denom) < 1e-6f)
  {
    return cuda::std::numeric_limits<fptype>::infinity();
  }
  auto d = dot(plane.position - rayOrigin, plane.normal) / denom;
  if (d < 0.f)
  {
    return cuda::std::numeric_limits<fptype>::infinity();
  }
  return d;
}
